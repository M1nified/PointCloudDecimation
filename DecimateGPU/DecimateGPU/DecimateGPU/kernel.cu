#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include "Point.h"

#include "TxtToBinConverter.h"

#include <stdio.h>
#include <fstream>
#include <string>
#include <iostream>

#define FILTER_BLOCKS_COUNT_Y 10000
#define FILTER_BLOCKS_CHUNK_SIZE 1000000

#define FILTER_HIT_SIZE 100000

#define CUDA_MEMCPY_FAIL_CHECK(cudaStatus) \
if (cudaStatus != hipSuccess) { \
	fprintf(stderr, "hipMemcpy failed!"); \
	goto Error; \
}
#define CUDA_MALLOC_FAIL_CHECK(cudaStatus) \
if (cudaStatus != hipSuccess) { \
	fprintf(stderr, "hipMalloc failed!"); \
	goto Error; \
}

typedef unsigned long long ull;
typedef short int si;

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void makeCubesKernel(const si d, const ull arrSize, bool * removedGpu, si * xGpu, si * yGpu, si * zGpu, si * bxGpu, si * byGpu, si * bzGpu, Point * pointsGpu)
{
	ull i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < arrSize)
	{
		xGpu[i] = pointsGpu[i].x % d;
		yGpu[i] = pointsGpu[i].y % d;
		zGpu[i] = pointsGpu[i].z % d;
		bxGpu[i] = pointsGpu[i].x / d;
		byGpu[i] = pointsGpu[i].y / d;
		bzGpu[i] = pointsGpu[i].z / d;
	}

}

__global__ void filterCubesKernel(const si d, const ull arrLength, bool * removedGpu, bool * siblingFound, si * xGpu, si * yGpu, si * zGpu, si * bxGpu, si * byGpu, si * bzGpu, Point * pointsGpu)
{
	ull i, j, jLimit;
	i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < arrLength && !siblingFound[i])
	{
		jLimit = (blockIdx.y + 1) * FILTER_BLOCKS_CHUNK_SIZE;
		//ull j = i + (blockIdx.y * FILTER_BLOCKS_COUNT_Y);
		//ull searchLimit = j + FILTER_BLOCKS_COUNT_Y;
		for (
			j = i + 1;
			j < arrLength
			&& j < jLimit
			//&& j < 100000
			/*&& !(bxGpu[i] == bxGpu[j]
				&& byGpu[i] == byGpu[j]
				&& bzGpu[i] == bzGpu[j])*/;
			j++)
		{
			if (bxGpu[i] == bzGpu[j]
				&& byGpu[i] == byGpu[j]
				&& bzGpu[i] == bzGpu[j])
			{
				break;
			}
		}
		if (j < arrLength 
			&& bxGpu[i] == bzGpu[j]
			&& byGpu[i] == byGpu[j]
			&& bzGpu[i] == bzGpu[j])
		{
			removedGpu[j] = true;
			siblingFound[i] = true;
		}
	}
}

//__global__ void filterCubesCmpKernel(const si d, const ull baseIndex, const si x, const si y, const si z, const si bx, const si by, const si bz, const ull arrLength, bool * removedGpu, si * xGpu, si * yGpu, si * zGpu, si * bxGpu, si * byGpu, si * bzGpu, Point * pointsGpu)
//{
//	ull i;
//	i = blockIdx.x*blockDim.x + threadIdx.x + baseIndex;
//	if (i < arrLength 
//		&& !removedGpu[i]
//		&& z == zGpu[i]
//		&& y == yGpu[i]
//		&& z == zGpu[i]
//		&& bx == bxGpu[i]
//		&& by == byGpu[i]
//		&& bz == bzGpu[i])
//	{
//		removedGpu[i] = true;
//	}
//}

__global__ void filterCubesCmpKernel(const si d, const ull baseIndex, si * xGpuTmp, si * yGpuTmp, si * zGpuTmp, si * bxGpuTmp, si * byGpuTmp, si * bzGpuTmp, const ull arrLength, bool * removedGpu, si * xGpu, si * yGpu, si * zGpu, si * bxGpu, si * byGpu, si * bzGpu, Point * pointsGpu)
{
	ull i, j;
	i = blockIdx.x*blockDim.x + threadIdx.x + baseIndex;
	if (i < arrLength && !removedGpu[i])
	{
		for (j = 0; j < FILTER_HIT_SIZE; j++)
		{
			if (zGpuTmp[j] == zGpu[i]
				&& yGpuTmp[j] == yGpu[i]
				&& zGpuTmp[j] == zGpu[i]
				&& bxGpuTmp[j] == bxGpu[i]
				&& byGpuTmp[j] == byGpu[i]
				&& bzGpuTmp[j] == bzGpu[i])
			{
				removedGpu[i] = true;
			}
		}
	}
}

int main()
{

	//auto converter = new TxtToBinConverter();
	//converter->SetTextFileName("D:\\decimate\\SONGA_BREEZE_L4.pts");
	//converter->SetBinFileName("K:\\SONGA_BREEZE_L4.bin");
	//converter->Convert();
	//delete converter;

	si d = 7;

	FILE * cloudFileBin = fopen("K:\SONGA_BREEZE_L4.bin", "rb");

	if (cloudFileBin == NULL)
	{
		return 1;
	}
	
	const ull sizeOfPoint = sizeof(struct Point);

	fseek(cloudFileBin, 0L, SEEK_END);
	ull binCloudFileSize = ftell(cloudFileBin);
	ull cloudCount = binCloudFileSize / sizeOfPoint;
	rewind(cloudFileBin);

	ull vramSizeInBytes = 1 * 1024 * 1024 * 1024;
	//vramSizeInBytes = 1920 * sizeOfPoint;
	ull buffCount = vramSizeInBytes / sizeOfPoint;
	buffCount = 1000000;
	ull buffSize = vramSizeInBytes - (vramSizeInBytes % sizeOfPoint);
	Point * cloudBuffer = (Point *)malloc(buffSize);

	ull pointsCount;
	hipError_t cudaStatus;

	bool * removed = (bool *)calloc(cloudCount, sizeof(bool));
	si * x = (si *)malloc(cloudCount * sizeof(si));
	si * y = (si *)malloc(cloudCount * sizeof(si));
	si * z = (si *)malloc(cloudCount * sizeof(si));
	si * bx = (si *)malloc(cloudCount * sizeof(si));
	si * by = (si *)malloc(cloudCount * sizeof(si));
	si * bz = (si *)malloc(cloudCount * sizeof(si));

	bool * removedGpu = false;
	bool * siblingFound = false;
	si * xGpu = 0;
	si * yGpu = 0;
	si * zGpu = 0;
	si * bxGpu = 0;
	si * byGpu = 0;
	si * bzGpu = 0;

	Point * pointsGpu = NULL;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	int maxBlockSize;
	int minGridSize;
	int gridSize;

	cudaStatus = hipMalloc((void**)&removedGpu, buffCount * sizeof(bool));
	CUDA_MALLOC_FAIL_CHECK(cudaStatus);
	cudaStatus = hipMalloc((void**)&siblingFound, buffCount * sizeof(bool));
	CUDA_MALLOC_FAIL_CHECK(cudaStatus);
	cudaStatus = hipMalloc((void**)&xGpu, buffCount * sizeof(si));
	CUDA_MALLOC_FAIL_CHECK(cudaStatus);
	cudaStatus = hipMalloc((void**)&yGpu, buffCount * sizeof(si));
	CUDA_MALLOC_FAIL_CHECK(cudaStatus);
	cudaStatus = hipMalloc((void**)&zGpu, buffCount * sizeof(si));
	CUDA_MALLOC_FAIL_CHECK(cudaStatus);
	cudaStatus = hipMalloc((void**)&bxGpu, buffCount * sizeof(si));
	CUDA_MALLOC_FAIL_CHECK(cudaStatus);
	cudaStatus = hipMalloc((void**)&byGpu, buffCount * sizeof(si));
	CUDA_MALLOC_FAIL_CHECK(cudaStatus);
	cudaStatus = hipMalloc((void**)&bzGpu, buffCount * sizeof(si));
	CUDA_MALLOC_FAIL_CHECK(cudaStatus);
	cudaStatus = hipMalloc((void**)&pointsGpu, buffCount * sizeOfPoint);
	CUDA_MALLOC_FAIL_CHECK(cudaStatus);

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	ull blockSize = 1024; // 1024

	for (ull i = 0; (pointsCount = fread(cloudBuffer, sizeOfPoint, buffCount, cloudFileBin)) != 0; i += pointsCount)
	{
		cudaStatus = hipMemcpy(pointsGpu, cloudBuffer, pointsCount * sizeOfPoint, hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		makeCubesKernel << <pointsCount / blockSize + 1, blockSize >> > (d, pointsCount, removedGpu, xGpu, yGpu, zGpu, bxGpu, byGpu, bzGpu, pointsGpu);

		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "makeCubesKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}

		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching makeCubesKernel!\n", cudaStatus);
			goto Error;
		}

		cudaStatus = hipMemcpy(&x[i], xGpu, pointsCount * sizeof(si), hipMemcpyDeviceToHost);
		CUDA_MEMCPY_FAIL_CHECK(cudaStatus);

		cudaStatus = hipMemcpy(&y[i], yGpu, pointsCount * sizeof(si), hipMemcpyDeviceToHost);
		CUDA_MEMCPY_FAIL_CHECK(cudaStatus);

		cudaStatus = hipMemcpy(&z[i], zGpu, pointsCount * sizeof(si), hipMemcpyDeviceToHost);
		CUDA_MEMCPY_FAIL_CHECK(cudaStatus);

		cudaStatus = hipMemcpy(&bx[i], bxGpu, pointsCount * sizeof(si), hipMemcpyDeviceToHost);
		CUDA_MEMCPY_FAIL_CHECK(cudaStatus);

		cudaStatus = hipMemcpy(&by[i], byGpu, pointsCount * sizeof(si), hipMemcpyDeviceToHost);
		CUDA_MEMCPY_FAIL_CHECK(cudaStatus);

		cudaStatus = hipMemcpy(&bz[i], bzGpu, pointsCount * sizeof(si), hipMemcpyDeviceToHost);
		CUDA_MEMCPY_FAIL_CHECK(cudaStatus);

	}

	// TEST

	dim3 gridDim(pointsCount / blockSize);
	//filterCubesKernel <<<gridDim, blockSize >>> (d, pointsCount, removedGpu, siblingFound, xGpu, yGpu, zGpu, bxGpu, byGpu, bzGpu, pointsGpu);
	
	si * xGpuTmp = 0;
	si * yGpuTmp = 0;
	si * zGpuTmp = 0;
	si * bxGpuTmp = 0;
	si * byGpuTmp = 0;
	si * bzGpuTmp = 0;

	cudaStatus = hipMalloc((void**)&xGpuTmp, FILTER_HIT_SIZE * sizeof(si));
	CUDA_MALLOC_FAIL_CHECK(cudaStatus);
	cudaStatus = hipMalloc((void**)&yGpuTmp, FILTER_HIT_SIZE * sizeof(si));
	CUDA_MALLOC_FAIL_CHECK(cudaStatus);
	cudaStatus = hipMalloc((void**)&zGpuTmp, FILTER_HIT_SIZE * sizeof(si));
	CUDA_MALLOC_FAIL_CHECK(cudaStatus);
	cudaStatus = hipMalloc((void**)&bxGpuTmp, FILTER_HIT_SIZE * sizeof(si));
	CUDA_MALLOC_FAIL_CHECK(cudaStatus);
	cudaStatus = hipMalloc((void**)&byGpuTmp, FILTER_HIT_SIZE * sizeof(si));
	CUDA_MALLOC_FAIL_CHECK(cudaStatus);
	cudaStatus = hipMalloc((void**)&bzGpuTmp, FILTER_HIT_SIZE * sizeof(si));
	CUDA_MALLOC_FAIL_CHECK(cudaStatus);

	si * xTmp = (si *)malloc(FILTER_HIT_SIZE * sizeof(si));
	si * yTmp = (si *)malloc(FILTER_HIT_SIZE * sizeof(si));
	si * zTmp = (si *)malloc(FILTER_HIT_SIZE * sizeof(si));
	si * bxTmp = (si *)malloc(FILTER_HIT_SIZE * sizeof(si));
	si * byTmp = (si *)malloc(FILTER_HIT_SIZE * sizeof(si));
	si * bzTmp = (si *)malloc(FILTER_HIT_SIZE * sizeof(si));

	for (ull i = 0; i < pointsCount; i+=FILTER_HIT_SIZE)
	{
		for (int j = 0; j < FILTER_HIT_SIZE; j++)
		{
			xTmp[j] = x[i + j];
			yTmp[j] = y[i + j];
			zTmp[j] = z[i + j];
			bxTmp[j] = bx[i + j];
			byTmp[j] = by[i + j];
			bzTmp[j] = bz[i + j];
		}
		cudaStatus = hipMemcpy(xTmp, xGpuTmp, FILTER_HIT_SIZE * sizeof(si), hipMemcpyDeviceToHost);
		CUDA_MEMCPY_FAIL_CHECK(cudaStatus);
		cudaStatus = hipMemcpy(yTmp, yGpuTmp, FILTER_HIT_SIZE * sizeof(si), hipMemcpyDeviceToHost);
		CUDA_MEMCPY_FAIL_CHECK(cudaStatus);
		cudaStatus = hipMemcpy(zTmp, zGpuTmp, FILTER_HIT_SIZE * sizeof(si), hipMemcpyDeviceToHost);
		CUDA_MEMCPY_FAIL_CHECK(cudaStatus);
		cudaStatus = hipMemcpy(bxTmp, bxGpuTmp, FILTER_HIT_SIZE * sizeof(si), hipMemcpyDeviceToHost);
		CUDA_MEMCPY_FAIL_CHECK(cudaStatus);
		cudaStatus = hipMemcpy(byTmp, byGpuTmp, FILTER_HIT_SIZE * sizeof(si), hipMemcpyDeviceToHost);
		CUDA_MEMCPY_FAIL_CHECK(cudaStatus);
		cudaStatus = hipMemcpy(bzTmp, bzGpuTmp, FILTER_HIT_SIZE * sizeof(si), hipMemcpyDeviceToHost);
		CUDA_MEMCPY_FAIL_CHECK(cudaStatus);

		ull gridSize;
		gridSize = (pointsCount - i) / blockSize;
		filterCubesCmpKernel << <gridSize, blockSize >> > (d, i, xGpuTmp, yGpuTmp, zGpuTmp, bxGpuTmp, byGpuTmp, bzGpuTmp, pointsCount, removedGpu, xGpu, yGpu, zGpu, bxGpu, byGpu, bzGpu, pointsGpu);

		//cudaStatus = hipGetLastError();
		//if (cudaStatus != hipSuccess) {
		//	fprintf(stderr, "makeCubesKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		//	goto Error;
		//}

	}

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "makeCubesKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching filterCubesKernel!\n", cudaStatus);
		goto Error;
	}

	cudaStatus = hipMemcpy(removed, removedGpu, pointsCount * sizeof(bool), hipMemcpyDeviceToHost);
	CUDA_MEMCPY_FAIL_CHECK(cudaStatus);


	ull removedCount = 0;
	for (ull i = 0; i < pointsCount; i++)
	{
		if (removed[i])
		{
			removedCount++;
		}
	}

	// TEST END

	return 0;

Error:
	return cudaStatus;
}

int main2()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
